#include "hip/hip_runtime.h"
#include <assert.h>

#include <iostream>

#include "computeTex.cuh"
#include "errorHandling.cuh"
#include "getActiveBlocks.cuh"
#include "minMaxReduction.cuh"

__device__ __inline__ float3 interpolate3(uint3 pos1, uint3 pos2, int w1,
                                          int w2) {
  return float3{(float)(pos1.x * w1 + pos2.x * w2) / (w1 + w2),
                (float)(pos1.y * w1 + pos2.y * w2) / (w1 + w2),
                (float)(pos1.z * w1 + pos2.z * w2) / (w1 + w2)};
}

__device__ void sampleVolume(uint3 pos, volatile int* shem,
                             hipTextureObject_t tex, float3* vertices) {
  int tid_block = threadIdx.x + blockDim.x * threadIdx.y +
                  blockDim.x * blockDim.y * threadIdx.z;

  // Neighbours in each direction
  int offsets[3] = {(threadIdx.x + 1) + blockDim.x * threadIdx.y +
                        blockDim.x * blockDim.y * threadIdx.z,
                    threadIdx.x + blockDim.x * (threadIdx.y + 1) +
                        blockDim.x * blockDim.y * threadIdx.z,
                    threadIdx.x + blockDim.x * threadIdx.y +
                        blockDim.x * blockDim.y * (threadIdx.z + 1)};

  uint3 next_vertices[3] = {uint3{pos.x + 1, pos.y, pos.z},
                            uint3{pos.x, pos.y + 1, pos.z},
                            uint3{pos.x, pos.y, pos.z + 1}};

  int next_voxels[3] = {0, 0, 0};

#pragma unroll
  // Check if vertex its out of boundaries
  for (size_t i = 0; i < 3; i++) {
    if (offsets[i] < blockDim.x * blockDim.y * blockDim.z)
      next_voxels[i] = shem[offsets[i]];
    else
      next_voxels[i] = tex3D<int>(tex, next_vertices[i].x, next_vertices[i].y,
                                  next_vertices[i].z);
  }

#pragma unroll
  for (size_t i = 0; i < 3; i++)
    vertices[i] =
        interpolate3(pos, next_vertices[i], shem[tid_block], next_voxels[i]);
}

__global__ void generateTris(hipTextureObject_t tex, int* activeBlocks,
                             int* numActiveBlocks) {
  int numBlk = *numActiveBlocks;
  int block_id = activeBlocks[blockIdx.x];
  int tid_block = threadIdx.x + blockDim.x * threadIdx.y +
                  blockDim.x * blockDim.y * threadIdx.z;

  int3 block_pos =
      int3{block_id % 16, (block_id / 16) % (16 * 16), block_id / (16 * 16)};
  uint3 pos = uint3{threadIdx.x + block_pos.x * blockDim.x,
                    threadIdx.y + block_pos.y * blockDim.y,
                    threadIdx.z + block_pos.z * blockDim.z};

  __shared__ int voxels[1024];
  voxels[tid_block] = tex3D<int>(tex, pos.x, pos.y, pos.z);
  __syncthreads();

  float3* vertices;
  vertices = new float3[3];
  sampleVolume(pos, voxels, tex, vertices);

  printf(
      "pos : %d %d %d : \n %f %f %f,  %f %f %f,  %f %f %f \n $$$$$$$$$$$$ \n",
      pos.x, pos.y, pos.z, vertices[0].x, vertices[0].y, vertices[0].z,
      vertices[1].x, vertices[1].y, vertices[1].z, vertices[2].x, vertices[2].y,
      vertices[2].z);
}

using namespace std;

int main() {
  int num_points_x = 128, num_points_y = 128, num_points_z = 128;
  int num_points = num_points_x * num_points_y * num_points_z;

  int* h_data = new int[num_points];

  int off_x[2] = {1, 0}, off_y[2] = {1, 0}, off_z[2] = {1, 0};
  int non_empty_cubes[6] = {0, 8, 16, 32};

  for (int x0 : non_empty_cubes)
    for (int i = 0; i < 2; i++)
      for (int j = 0; j < 2; j++)
        for (int k = 0; k < 2; k++)
          h_data[(x0 + off_x[i]) + num_points_x * (off_y[j]) +
                 num_points_x * num_points_y * (off_z[k])] = x0 + 20;

  ComputeTex ct(h_data, num_points_x, num_points_y, num_points_z);

  int n_x = (num_points_x - 1), n_y = (num_points_y - 1),
      n_z = (num_points_y - 1);
  int n = n_x * n_y * n_z;

  dim3 block_size = {8, 8, 8};
  dim3 grid_size = {(n_x + block_size.x - 1) / block_size.x,
                    (n_y + block_size.y - 1) / block_size.y,
                    (n_z + block_size.z - 1) / block_size.z};
  int num_blocks = grid_size.x * grid_size.y * grid_size.z;

  int2* h_blockMinMax = new int2[num_blocks];
  int2* g_blockMinMax;
  int* g_h_activeBlkNum;
  int* g_numActiveBlocks;

  hipMalloc(&g_blockMinMax, num_blocks * sizeof(int2));
  hipMallocManaged(&g_h_activeBlkNum, num_blocks * sizeof(int));
  hipMalloc(&g_numActiveBlocks, num_blocks * sizeof(int));

  for (int i = 0; i < num_blocks; i++) g_h_activeBlkNum[i] = -1;

  blockReduceMinMax<<<grid_size, block_size>>>(ct.texObj, n, g_blockMinMax);

  int block_size2 = 128;
  int grid_size2 = (num_blocks + block_size2 - 1) / block_size2;
  getActiveBlocks<<<grid_size2, block_size2>>>(
      g_blockMinMax, num_blocks, g_h_activeBlkNum, g_numActiveBlocks);

  int* d_numActiveBlk = g_numActiveBlocks + block_size2 - 1;

  hipDeviceSynchronize();

  int numActiveBlk = 0;
  hipMemcpy(&numActiveBlk, g_numActiveBlocks + block_size2 - 1, sizeof(int),
             hipMemcpyDeviceToHost);

  hipMemcpy(h_blockMinMax, g_blockMinMax, num_blocks * sizeof(int2),
             hipMemcpyDeviceToHost);

  dim3 block_size3 = block_size;
  int num_blocks3 = block_size3.x * block_size.y + block_size.z;
  dim3 grid_size3 = {numActiveBlk};

  generateTris<<<grid_size3, block_size3>>>(ct.texObj, g_h_activeBlkNum,
                                            d_numActiveBlk);

  for (int i = 0; i < 8; i++)
    cout << g_h_activeBlkNum[i] << " " << h_blockMinMax[i].x << " "
         << h_blockMinMax[i].y << " " << i << endl;
}
