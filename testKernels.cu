#include "hip/hip_runtime.h"

#include <assert.h>

#include <iostream>

#include "getActiveBlocks.cuh"
#include "minMaxReduction.cuh"

using namespace std;

int main() {
  int n_x = 128, n_y = 128, n_z = 128;
  int n = n_x * n_y * n_z;

  dim3 block_size = {8, 8, 8};
  dim3 grid_size = {(n_x + block_size.x - 1) / block_size.x,
                    (n_y + block_size.y - 1) / block_size.y,
                    (n_z + block_size.z - 1) / block_size.z};
  int num_blocks = grid_size.x * grid_size.y * grid_size.z;

  int* h_data = new int[n];
  int2* h_blockMinMax = new int2[num_blocks];
  int* g_data;
  int2* g_blockMinMax;
  int* g_h_activeBlkNum;
  int* g_numActiveBlocks;

  hipMalloc(&g_data, n * sizeof(int));
  hipMalloc(&g_blockMinMax, num_blocks * sizeof(int2));
  hipMallocManaged(&g_h_activeBlkNum, num_blocks * sizeof(int));
  hipMalloc(&g_numActiveBlocks, num_blocks * sizeof(int));

  for (int i = 0; i < n; i++) h_data[i] = 0;

  for (int i = 0; i < num_blocks; i++) g_h_activeBlkNum[i] = -1;

  int off_x[2] = {1, 0}, off_y[2] = {1, 0}, off_z[2] = {1, 0};
  int non_empty_cubes[6] = {0, 8, 512, 1024, 1032, 16384};

  for (int x0 : non_empty_cubes)
    for (int i = 0; i < 2; i++)
      for (int j = 0; j < 2; j++)
        for (int k = 0; k < 2; k++)
          h_data[(x0 + off_x[i]) + 8 * (off_y[j]) + 8 * 8 * (off_z[k])] =
              x0 + 100;

  hipMemcpy(g_data, h_data, n * sizeof(int), hipMemcpyHostToDevice);

  blockReduceMinMax<<<grid_size, block_size>>>(g_data, n_x, n_y, n_z,
                                               g_blockMinMax);

  int block_size2 = 128;
  int grid_size2 = (num_blocks + block_size2 - 1) / block_size2;
  getActiveBlocks<<<grid_size2, block_size2>>>(
      g_blockMinMax, num_blocks, g_h_activeBlkNum, g_numActiveBlocks);

  hipDeviceSynchronize();

  hipMemcpy(h_blockMinMax, g_blockMinMax, num_blocks * sizeof(int2),
             hipMemcpyDeviceToHost);

  for (int i = 0; i < 257; i++)
    cout << g_h_activeBlkNum[i] << " " << h_blockMinMax[i].x << " "
         << h_blockMinMax[i].y << " " << i << endl;
}
