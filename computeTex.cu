#include "computeTex.cuh"

ComputeTex::ComputeTex() {
  nx = 0;
  ny = 0;
  nz = 0;
}

ComputeTex ::ComputeTex(int *h_data, int nx, int ny, int nz) {
  // Volume size
  const hipExtent volumeSize = make_hipExtent(nx, ny, nz);

  // Allocate CUDA array in device memory
  hipChannelFormatDesc channeldesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
  hipMalloc3DArray(&cuArray, &channeldesc, volumeSize);

  // copy data to 3d array
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr =
      make_hipPitchedPtr((void *)h_data, volumeSize.width * sizeof(int),
                          volumeSize.width, volumeSize.height);
  copyParams.dstArray = cuArray;
  copyParams.extent = volumeSize;
  copyParams.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&copyParams);

  // Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray;

  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.addressMode[2] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  // Create texture object
  texObj = 0;
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
}

ComputeTex ::~ComputeTex() {
  if (nx == 0 && ny == 0 && nz == 0) return;
  // Destroy texture object
  hipDestroyTextureObject(texObj);
  // Free device memory
  hipFreeArray(cuArray);
}

