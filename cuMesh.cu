#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <cuda_gl_interop.h>

#include <iostream>

#include "cuMesh.cuh"

using namespace std;

cuMesh::cuMesh(float3* h_vertices, uint3* h_indices, size_t n_vertices,
               size_t n_indices) {
  glGenBuffers(1, &VBO);
  glGenBuffers(1, &EBO);
  glGenVertexArrays(1, &VAO);

  glBindVertexArray(VAO);

  glBindBuffer(GL_ARRAY_BUFFER, VBO);
  uint size_VBO = n_vertices * sizeof(float3);
  glBufferData(GL_ARRAY_BUFFER, size_VBO, 0, GL_DYNAMIC_DRAW);

  glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
  uint size_EBO = n_indices * sizeof(int3);
  glBufferData(GL_ELEMENT_ARRAY_BUFFER, size_EBO, 0, GL_DYNAMIC_DRAW);

  mapVBO();
  mapEBO();

  hipMemcpy(d_vertices, h_vertices, n_vertices * sizeof(float3),
             hipMemcpyHostToDevice);
  hipMemcpy(d_indices, h_indices, n_indices * sizeof(uint3),
             hipMemcpyHostToDevice);
}

cuMesh::cuMesh() {}

cuMesh::~cuMesh() { }

void cuMesh::mapVBO() {
  hipGraphicsGLRegisterBuffer(&positionsVBO_CUDA, VBO,
                               cudaGraphicsMapFlagsWriteDiscard);
  hipGraphicsMapResources(1, &positionsVBO_CUDA, 0);
  hipGraphicsResourceGetMappedPointer((void**)&d_vertices, nullptr,
                                       positionsVBO_CUDA);
}

void cuMesh::mapEBO() {
  hipGraphicsGLRegisterBuffer(&indicesEBO_CUDA, EBO,
                               cudaGraphicsMapFlagsWriteDiscard);
  hipGraphicsMapResources(1, &indicesEBO_CUDA, 0);
  hipGraphicsResourceGetMappedPointer((void**)&d_indices, nullptr,
                                       indicesEBO_CUDA);
}

void cuMesh::deleteVBO_CUDA() {
  hipGraphicsUnmapResources(1, &indicesEBO_CUDA, 0);
  hipGraphicsUnregisterResource(indicesEBO_CUDA);

  hipGraphicsUnmapResources(1, &positionsVBO_CUDA, 0);
  hipGraphicsUnregisterResource(positionsVBO_CUDA);

  glDeleteBuffers(1, &VBO);
  glDeleteBuffers(1, &EBO);
  glDeleteVertexArrays(1, &VAO);
}

