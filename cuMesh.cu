#include "hip/hip_runtime.h"
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include <iostream>

#include "cuMesh.cuh"

using namespace std;

__global__ void deformVertices(float3* pos, int n) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x < n) {
    pos[x].x *= 1.0f;
    pos[x].y *= 1.0f;
    pos[x].z *= 1.0f;
  }
}

cuMesh::cuMesh(unsigned int VBO, unsigned int EBO) {
  mapVBO(VBO);
  mapEBO(EBO);
}

cuMesh::cuMesh() {}

cuMesh::~cuMesh() { deleteVBO_CUDA(); }

void cuMesh::mapVBO(unsigned int VBO) {
  vertices_g = nullptr;
  hipGraphicsGLRegisterBuffer(&positionsVBO_CUDA, VBO,
                               cudaGraphicsMapFlagsWriteDiscard);
  hipGraphicsMapResources(1, &positionsVBO_CUDA, 0);
}

void cuMesh::mapEBO(unsigned int EBO) {
  indices_g = nullptr;
  hipGraphicsGLRegisterBuffer(&indicesEBO_CUDA, EBO,
                               cudaGraphicsMapFlagsWriteDiscard);
  hipGraphicsMapResources(1, &indicesEBO_CUDA, 0);
}

void cuMesh::deleteVBO_CUDA() {
  hipGraphicsUnmapResources(1, &indicesEBO_CUDA, 0);
  hipGraphicsUnregisterResource(indicesEBO_CUDA);
  indices_g = nullptr;

  hipGraphicsUnmapResources(1, &positionsVBO_CUDA, 0);
  hipGraphicsUnregisterResource(positionsVBO_CUDA);
  vertices_g = nullptr;
}

void cuMesh::callKernel() {
  size_t num_bytes_v;
  hipGraphicsResourceGetMappedPointer((void**)&vertices_g, &num_bytes_v,
                                       positionsVBO_CUDA);
  size_t num_vertices = num_bytes_v / (sizeof(float3));
  vertices_h = (float3*)malloc(num_bytes_v);
  hipMemcpy(vertices_h, vertices_g, num_bytes_v, hipMemcpyDeviceToHost);

  size_t num_bytes_i;
  hipGraphicsResourceGetMappedPointer((void**)&indices_g, &num_bytes_i,
                                       indicesEBO_CUDA);
  size_t num_tri = num_bytes_i / (sizeof(int3));
  indices_h = (int3*)malloc(num_bytes_i);
  hipMemcpy(indices_h, indices_g, num_bytes_i, hipMemcpyDeviceToHost);

  dim3 dimBlock(16, 1, 1);
  dim3 dimGrid(ceil((float)num_vertices / dimBlock.x), 1, 1);

  deformVertices<<<dimGrid, dimBlock>>>(vertices_g, num_vertices);
}
