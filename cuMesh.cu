#include "hip/hip_runtime.h"
#include <cuda_gl_interop.h>
#include <glad/glad.h>

#include "cuMesh.cuh"

cuMesh::cuMesh(float3* h_vertices, int3* h_indices, size_t n_vertices,
               size_t n_indices) {
  hipMemcpy(d_vertices, h_vertices, n_vertices * sizeof(float3),
             hipMemcpyHostToDevice);
  hipMemcpy(d_indices, h_indices, n_indices * sizeof(int3),
             hipMemcpyHostToDevice);

  glGenBuffers(1, &VBO);
  glGenBuffers(1, &EBO);

  mapVBO();
  mapEBO();
}

cuMesh::cuMesh() {}

cuMesh::~cuMesh() { deleteVBO_CUDA(); }

void cuMesh::mapVBO() {
  hipGraphicsGLRegisterBuffer(&positionsVBO_CUDA, VBO,
                               cudaGraphicsMapFlagsWriteDiscard);
  hipGraphicsMapResources(1, &positionsVBO_CUDA, 0);
}

void cuMesh::mapEBO() {
  hipGraphicsGLRegisterBuffer(&indicesEBO_CUDA, EBO,
                               cudaGraphicsMapFlagsWriteDiscard);
  hipGraphicsMapResources(1, &indicesEBO_CUDA, 0);
}

void cuMesh::deleteVBO_CUDA() {
  hipFree(d_indices);
  hipGraphicsUnmapResources(1, &indicesEBO_CUDA, 0);
  hipGraphicsUnregisterResource(indicesEBO_CUDA);

  hipFree(d_vertices);
  hipGraphicsUnmapResources(1, &positionsVBO_CUDA, 0);
  hipGraphicsUnregisterResource(positionsVBO_CUDA);

  glDeleteBuffers(1, &VBO);
  glDeleteBuffers(1, &EBO);
}

