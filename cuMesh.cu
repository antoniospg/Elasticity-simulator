#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <cuda_gl_interop.h>

#include <iostream>

#include "cuMesh.cuh"

using namespace std;

cuMesh::cuMesh(float3* vertices, int3* indices, size_t n_vertices,
               size_t n_indices, bool device_pointers) {
  glGenBuffers(1, &VBO);
  glGenBuffers(1, &EBO);
  glGenVertexArrays(1, &VAO);

  glBindVertexArray(VAO);

  glBindBuffer(GL_ARRAY_BUFFER, VBO);
  uint size_VBO = n_vertices * sizeof(float3);
  glBufferData(GL_ARRAY_BUFFER, size_VBO, 0, GL_DYNAMIC_DRAW);

  glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
  uint size_EBO = n_indices * sizeof(int3);
  glBufferData(GL_ELEMENT_ARRAY_BUFFER, size_EBO, 0, GL_DYNAMIC_DRAW);

  mapVBO();
  mapEBO();

  if (device_pointers) {
    hipMemcpy(d_vertices, vertices, n_vertices * sizeof(float3),
               hipMemcpyDeviceToDevice);
    hipMemcpy(d_indices, indices, n_indices * sizeof(uint3),
               hipMemcpyDeviceToDevice);
    hipDeviceSynchronize();
  } else {
    hipMemcpy(d_vertices, vertices, n_vertices * sizeof(float3),
               hipMemcpyHostToDevice);
    hipMemcpy(d_indices, indices, n_indices * sizeof(uint3),
               hipMemcpyHostToDevice);
  }
}

cuMesh::cuMesh() {}

cuMesh::~cuMesh() {}

void cuMesh::mapVBO() {
  hipGraphicsGLRegisterBuffer(&positionsVBO_CUDA, VBO,
                               cudaGraphicsMapFlagsWriteDiscard);
  hipGraphicsMapResources(1, &positionsVBO_CUDA, 0);
  hipGraphicsResourceGetMappedPointer((void**)&d_vertices, nullptr,
                                       positionsVBO_CUDA);
}

void cuMesh::mapEBO() {
  hipGraphicsGLRegisterBuffer(&indicesEBO_CUDA, EBO,
                               cudaGraphicsMapFlagsWriteDiscard);
  hipGraphicsMapResources(1, &indicesEBO_CUDA, 0);
  hipGraphicsResourceGetMappedPointer((void**)&d_indices, nullptr,
                                       indicesEBO_CUDA);
}

void cuMesh::deleteVBO_CUDA() {
  hipGraphicsUnmapResources(1, &indicesEBO_CUDA, 0);
  hipGraphicsUnregisterResource(indicesEBO_CUDA);

  hipGraphicsUnmapResources(1, &positionsVBO_CUDA, 0);
  hipGraphicsUnregisterResource(positionsVBO_CUDA);

  glDeleteBuffers(1, &VBO);
  glDeleteBuffers(1, &EBO);
  glDeleteVertexArrays(1, &VAO);
}
