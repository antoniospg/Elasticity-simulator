#include "hip/hip_runtime.h"
#include <iostream>

#include "computeTex.cuh"
using namespace std;

ComputeTex::ComputeTex() {
  nx = 0;
  ny = 0;
  nz = 0;
}

__global__ void texKernels::getNormals(hipTextureObject_t tex, int nx, int ny,
                                       int nz, float4* ans) {
  int3 pos = {threadIdx.x + blockDim.x * blockIdx.x,
              threadIdx.y + blockDim.y * blockIdx.y,
              threadIdx.z + blockDim.z * blockIdx.z};
  int id = pos.x + nx * pos.y + nx * ny * pos.z;

  float4 dv = {0, 0, 0, 0};

  dv.x = tex3D<int>(tex, pos.x + 1, pos.y, pos.z) -
         tex3D<int>(tex, pos.x - 1, pos.y, pos.z);
  dv.x /= 2;

  dv.y = tex3D<int>(tex, pos.x, pos.y + 1, pos.z) -
         tex3D<int>(tex, pos.x, pos.y - 1, pos.z);
  dv.y /= 2;

  dv.z = tex3D<int>(tex, pos.x, pos.y, pos.z + 1) -
         tex3D<int>(tex, pos.x, pos.y, pos.z - 1);
  dv.z /= 2;

  dv.w = 1;

  ans[id] = dv;
}

ComputeTex ::ComputeTex(int* h_data, int nx, int ny, int nz) {
  // Volume size
  const hipExtent volumeSize = make_hipExtent(nx, ny, nz);

  // Allocate CUDA array in device memory
  hipChannelFormatDesc channeldesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindSigned);
  hipMalloc3DArray(&cuArray, &channeldesc, volumeSize);

  // copy data to 3d array
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr =
      make_hipPitchedPtr((void*)h_data, volumeSize.width * sizeof(int),
                          volumeSize.width, volumeSize.height);
  copyParams.dstArray = cuArray;
  copyParams.extent = volumeSize;
  copyParams.kind = hipMemcpyHostToDevice;
  hipMemcpy3D(&copyParams);

  // Specify texture
  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray;

  // Specify texture object parameters
  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeBorder;
  texDesc.addressMode[1] = hipAddressModeBorder;
  texDesc.addressMode[2] = hipAddressModeBorder;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  // Create texture object
  texObj = 0;
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

  // Create texture with normals
  float4* d_normal_data;
  hipMalloc(&d_normal_data, nx * ny * nz * sizeof(float4));

  dim3 block_size = {8, 8, 8};
  dim3 grid_size;
  grid_size.x = (nx + block_size.x - 1) / block_size.x;
  grid_size.y = (ny + block_size.y - 1) / block_size.y;
  grid_size.z = (nz + block_size.z - 1) / block_size.z;

  texKernels::getNormals<<<grid_size, block_size>>>(texObj, nx, ny, nz,
                                                    d_normal_data);

  // NORMAL TEXTURE
  // Allocate CUDA array in device memory
  channeldesc =
      hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindSigned);
  hipMalloc3DArray(&cuArrayNormal, &channeldesc, volumeSize);

  // copy data to 3d array
  copyParams = {0};
  copyParams.srcPtr = make_hipPitchedPtr((void*)d_normal_data,
                                          volumeSize.width * sizeof(float4),
                                          volumeSize.width, volumeSize.height);
  copyParams.dstArray = cuArrayNormal;
  copyParams.extent = volumeSize;
  copyParams.kind = hipMemcpyDeviceToDevice;
  hipMemcpy3D(&copyParams);

  // Specify texture
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArrayNormal;

  // Specify texture object parameters
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.addressMode[2] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 0;

  texObjNormal = 0;
  hipCreateTextureObject(&texObjNormal, &resDesc, &texDesc, NULL);
  hipFree(d_normal_data);
}

ComputeTex ::~ComputeTex() {
  if (nx == 0 && ny == 0 && nz == 0) return;
  // Destroy texture object
  hipDestroyTextureObject(texObj);
  hipDestroyTextureObject(texObjNormal);
  // Free device memory
  hipFreeArray(cuArray);
  hipFreeArray(cuArrayNormal);
}

