
#include <hip/hip_runtime.h>
#include <assert.h>

#include <algorithm>
#include <iostream>

#define WP_SIZE 32

using namespace std;

__device__ __inline__ int2 warpReduceMinMax(int2 val) {
  for (int offset = WP_SIZE / 2; offset > 0; offset /= 2) {
    val.x = max(val.x, __shfl_down_sync(-1, val.x, offset));
    val.y = min(val.y, __shfl_down_sync(-1, val.y, offset));
  }
  return val;
}
__global__ void blockReduceMinMax(int* g_data, int n_x, int n_y, int n_z,
                                  int2* g_ans) {
  int tid_block = (threadIdx.z * blockDim.y * blockDim.x +
                   threadIdx.y * blockDim.x + threadIdx.x);
  int lane = tid_block % WP_SIZE;
  int bid = (blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x +
             blockIdx.x);
  int wid = tid_block / WP_SIZE;
  int tid = tid_block + (blockDim.x * blockDim.y * blockDim.z) * bid;

  __shared__ int2 warpAns[32];

  int2 val = {g_data[tid], g_data[tid]};
  val = warpReduceMinMax(val);

  if (lane == 0) warpAns[wid] = val;

  __syncthreads();

  val = (tid_block < blockDim.x * blockDim.y * blockDim.z / WP_SIZE)
            ? warpAns[lane]
            : (int2){0, 1e9 + 1};

  if (wid == 0) val = warpReduceMinMax(val);
  if (tid_block == 0) g_ans[bid] = val;
}

int main() {
  int n_x = 128, n_y = 128, n_z = 128;
  int n = n_x * n_y * n_z;

  dim3 block_size = {8, 8, 8};
  dim3 grid_size = {(n_x + block_size.x - 1) / block_size.x,
                    (n_y + block_size.y - 1) / block_size.y,
                    (n_z + block_size.z - 1) / block_size.z};
  int num_blocks = grid_size.x * grid_size.y * grid_size.z;

  int* h_data = new int[n];
  int2* h_ans = new int2[num_blocks];
  for (int i = 0; i < n; i++) h_data[i] = 0;

  int off_x[2] = {1, 0}, off_y[2] = {1, 0}, off_z[2] = {1, 0};
  int non_empty_cubes[6] = {0, 8, 512, 1024, 1032, 16384};
  for (int x0 : non_empty_cubes)
    for (int i = 0; i < 2; i++)
      for (int j = 0; j < 2; j++)
        for (int k = 0; k < 2; k++)
          h_data[(x0 + off_x[i]) + 8 * (off_y[j]) +
                 8 * 8 * (off_z[k])] = x0 + 2;

  int* g_data;
  int2* g_ans;
  hipMalloc(&g_data, n * sizeof(int));
  hipMalloc(&g_ans, num_blocks * sizeof(int2));
  hipMemcpy(g_data, h_data, n * sizeof(int), hipMemcpyHostToDevice);

  blockReduceMinMax<<<grid_size, block_size>>>(g_data, n_x, n_y, n_z, g_ans);
  hipMemcpy(h_ans, g_ans, num_blocks * sizeof(int2), hipMemcpyDeviceToHost);

  for (int i = 0; i < 64; i++)
    cout << h_ans[i].x << " " << h_ans[i].y << " " << i << endl;
}
