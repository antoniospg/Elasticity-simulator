#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>

#include <iostream>

#include "constants.h"
#include "genTriangles.cuh"

#define INF 1e12
using namespace std;

namespace cg = cooperative_groups;

__constant__ int d_neighbourMappingTable[12][4];
__constant__ int d_edgeTable[256];
__constant__ int d_triTable[256][16];
__constant__ int d_isoVal;
__device__ int my_block_count3_0 = 0;
__device__ int my_block_count3_1 = 0;

__device__ int genTriangles::getCubeidx(int3 pos, volatile int* shem) {
  int tid_block = threadIdx.x + blockDim.x * threadIdx.y +
                  blockDim.x * blockDim.y * threadIdx.z;

  pos = {(int)threadIdx.x, (int)threadIdx.y, (int)threadIdx.z};
  // Neighbours in each direction
  int offsets[8] = {threadIdx.x + blockDim.x * threadIdx.y +
                        blockDim.x * blockDim.y * threadIdx.z,
                    (threadIdx.x + 1) + blockDim.x * threadIdx.y +
                        blockDim.x * blockDim.y * threadIdx.z,
                    (threadIdx.x + 1) + blockDim.x * threadIdx.y +
                        blockDim.x * blockDim.y * (threadIdx.z + 1),
                    threadIdx.x + blockDim.x * threadIdx.y +
                        blockDim.x * blockDim.y * (threadIdx.z + 1),
                    threadIdx.x + blockDim.x * (threadIdx.y - 1) +
                        blockDim.x * blockDim.y * threadIdx.z,
                    (threadIdx.x + 1) + blockDim.x * (threadIdx.y - 1) +
                        blockDim.x * blockDim.y * threadIdx.z,
                    (threadIdx.x + 1) + blockDim.x * (threadIdx.y - 1) +
                        blockDim.x * blockDim.y * (threadIdx.z + 1),
                    threadIdx.x + blockDim.x * (threadIdx.y - 1) +
                        blockDim.x * blockDim.y * (threadIdx.z + 1)};
  int3 pos_offset[8] = {{pos.x, pos.y, pos.z},
                        {pos.x + 1, pos.y, pos.z},
                        {pos.x + 1, pos.y, pos.z + 1},
                        {pos.x, pos.y, pos.z + 1},
                        {pos.x, pos.y - 1, pos.z},
                        {pos.x + 1, pos.y - 1, pos.z},
                        {pos.x + 1, pos.y - 1, pos.z + 1},
                        {pos.x, pos.y - 1, pos.z + 1}};

  int cubeindex = 0, increment = 1;
  for (size_t i = 0; i < 8; i++) {
    if (pos_offset[i].x < blockDim.x && pos_offset[i].y >= 0 &&
        pos_offset[i].z < blockDim.z && shem[offsets[i]] < d_isoVal)
      cubeindex |= increment;
    else if (pos_offset[i].x >= blockDim.x || pos_offset[i].y < 0 ||
             pos_offset[i].z >= blockDim.z)
      cubeindex |= increment;
    increment *= 2;
  }

  return cubeindex;
}

__device__ __inline__ float3 genTriangles::lerpVertex(int3 pos1, int3 pos2,
                                                      int v1, int v2) {
  float3 vertex;
  float w = ((float)(d_isoVal - v1)) / (v2 - v1);

  vertex.x = pos1.x + w * (pos2.x - pos1.x);
  vertex.y = pos1.y + w * (pos2.y - pos1.y);
  vertex.z = pos1.z + w * (pos2.z - pos1.z);

  return vertex;
}

__device__ bool3 genTriangles::getVertex(int3 pos, bool3& active_edges,
                                         volatile int* shem, float3* vertices) {
  int offset[4] = {(threadIdx.x) + blockDim.x * (threadIdx.y) +
                       blockDim.x * blockDim.y * (threadIdx.z),
                   (threadIdx.x + 1) + blockDim.x * (threadIdx.y) +
                       blockDim.x * blockDim.y * (threadIdx.z),
                   (threadIdx.x) + blockDim.x * (threadIdx.y - 1) +
                       blockDim.x * blockDim.y * (threadIdx.z),
                   (threadIdx.x) + blockDim.x * (threadIdx.y) +
                       blockDim.x * blockDim.y * (threadIdx.z + 1)};

  int3 check_offset[3] = {{threadIdx.x + 1, threadIdx.y, threadIdx.z},
                          {threadIdx.x, threadIdx.y - 1, threadIdx.z},
                          {threadIdx.x, threadIdx.y, threadIdx.z + 1}};

  int3 pos_neigh[3] = {int3{pos.x + 1, pos.y, pos.z},
                       int3{pos.x, pos.y - 1, pos.z},
                       int3{pos.x, pos.y, pos.z + 1}};

  bool active_edges_array[3] = {active_edges.x, active_edges.y, active_edges.z};

  for (size_t i = 0; i < 3; i++) {
    if (check_offset[i].x < blockDim.x && check_offset[i].y >= 0 &&
        check_offset[i].z < blockDim.z && active_edges_array[i]) {
      vertices[i] =
          lerpVertex(pos, pos_neigh[i], shem[offset[0]], shem[offset[i + 1]]);

    } else if ((check_offset[i].x >= blockDim.x || check_offset[i].y < 0 ||
                check_offset[i].z >= blockDim.z) &&
               active_edges_array[i]) {
      vertices[i] = lerpVertex(pos, pos_neigh[i], 0, INF);
    }
  }
}

__device__ __inline__ int genTriangles::warpReduceScan(int val, int laneid) {
#pragma unroll
  for (int offset = 1; offset < WP_SIZE; offset *= 2) {
    int y = __shfl_up_sync(0xffffffff, val, offset);
    if (laneid >= offset) val += y;
  }
  return val;
}

__device__ int genTriangles::getVertexOffset(int nums) {
  int tid_block = (threadIdx.z * blockDim.y * blockDim.x +
                   threadIdx.y * blockDim.x + threadIdx.x);
  int bid = (blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x +
             blockIdx.x);
  int tid = tid_block + (blockDim.x * blockDim.y * blockDim.z) * bid;

  int lane = tid_block % WP_SIZE;
  int wid = tid_block / WP_SIZE;

  __shared__ int shem[32];
  nums = warpReduceScan(nums, lane);

  if (lane == 31) shem[wid] = nums;
  __syncthreads();

  int val = (tid_block < blockDim.x * blockDim.y * blockDim.z / WP_SIZE)
                ? shem[lane]
                : 0;
  if (wid == 0) {
    val = warpReduceScan(val, lane);
    shem[lane] = val;
  }
  __syncthreads();

  nums += (wid > 0) ? shem[wid - 1] : 0;

  return nums;
}

__device__ int genTriangles::borrowVertex(int3 pos, int edge,
                                          volatile int3* shem) {
  int tid_block = threadIdx.x + blockDim.x * threadIdx.y +
                  blockDim.x * blockDim.y * threadIdx.z;

  int offset[4] = {
      d_neighbourMappingTable[edge][0], d_neighbourMappingTable[edge][1],
      d_neighbourMappingTable[edge][2], d_neighbourMappingTable[edge][3]};

  int shem_id = (threadIdx.x + offset[0]) +
                blockDim.x * (threadIdx.y - offset[2]) +
                blockDim.x * blockDim.y * (threadIdx.z + offset[1]);
  int3 offset_pos = {threadIdx.x + offset[0], threadIdx.y - offset[2],
                     threadIdx.z + offset[1]};

  if (offset_pos.x < blockDim.x && offset_pos.y >= 0 &&
      offset_pos.z < blockDim.z) {
    if (offset[3] == 0)
      return shem[shem_id].x;
    else if (offset[3] == 1)
      return shem[shem_id].z;
    else if (offset[3] == 2)
      return shem[shem_id].y;
    else
      return -1;
  } else
    return -1;
}

__global__ void genTriangles::generateTris(hipTextureObject_t tex,
                                           int* activeBlocks,
                                           int* numActiveBlocks, dim3 grid_size,
                                           int* block_vertex_offset,
                                           int* block_index_offset,
                                           float3* vertices, int3* indices) {
  uint numBlk = *numActiveBlocks;
  int block_id = activeBlocks[blockIdx.x];
  int tid_block = threadIdx.x + blockDim.x * threadIdx.y +
                  blockDim.x * blockDim.y * threadIdx.z;

  int3 block_pos = int3{block_id % (int)grid_size.x,
                        (block_id / (int)grid_size.x) % ((int)grid_size.y),
                        block_id / ((int)grid_size.x * (int)grid_size.y)};
  int3 pos = {threadIdx.x + block_pos.x * blockDim.x,
              threadIdx.y + block_pos.y * blockDim.y,
              threadIdx.z + block_pos.z * blockDim.z};

  __shared__ unsigned int my_blockId;
  if (tid_block == 0) my_blockId = atomicAdd(&my_block_count3_0, 1);
  __syncthreads();

  __shared__ int voxels[1024];
  voxels[tid_block] = tex3D<int>(tex, pos.x, pos.y, pos.z);
  __syncthreads();

  int cube_idx = getCubeidx(pos, voxels);
  float3 vertices_local[3];
  vertices_local[0] = float3{0.0, 0.0, 0.0};
  vertices_local[1] = float3{0.0, 0.0, 0.0};
  vertices_local[2] = float3{0.0, 0.0, 0.0};

  int active_hash = d_edgeTable[cube_idx];
  bool3 active_edge;
  active_edge.x = (active_hash & 1) == 1;
  active_edge.y = (active_hash & 256) == 256;
  active_edge.z = (active_hash & 8) == 8;

  getVertex(pos, active_edge, voxels, vertices_local);

  int vertex_count = 0;
  if (active_edge.x) vertex_count++;
  if (active_edge.y) vertex_count++;
  if (active_edge.z) vertex_count++;

  int vertex_offset = getVertexOffset(vertex_count);
  int vertex_offset_next = vertex_offset;
  vertex_offset -= vertex_count;

  int3 vertex_block_id;
  vertex_block_id.x = active_edge.x + vertex_offset;
  vertex_block_id.y = active_edge.y + vertex_block_id.x;
  vertex_block_id.z = active_edge.z + vertex_block_id.y;

  __shared__ int3 vertices_block_id[1024];
  vertices_block_id[tid_block] = vertex_block_id;
  __syncthreads();

  int tris[18];
  int num_tris = 0;
  for (size_t i = 0; i < 18 && d_triTable[cube_idx][i] != -1; i++) {
    int tri_idx = borrowVertex(pos, d_triTable[cube_idx][i], vertices_block_id);
    tris[num_tris] = tri_idx;
    num_tris++;
  }
  num_tris /= 3;

  int index_offset = getVertexOffset(num_tris);
  int index_offset_next = index_offset;
  index_offset -= num_tris;

  // Global Offset
  if (tid_block == blockDim.x * blockDim.y * blockDim.z - 1) {
    int2 block_sum = {vertex_offset_next, index_offset_next};

    for (int i = blockIdx.x + 1; i < gridDim.x + 1; i++) {
      atomicAdd(block_vertex_offset + i, block_sum.x);
      atomicAdd(block_index_offset + i, block_sum.y);
      __threadfence();
    }

    atomicAdd(&my_block_count3_1, 1);
    do {
    } while (atomicAdd(&my_block_count3_1, 0) != my_block_count3_0);
  }
  __syncthreads();

  // Write vertices to global memory
  int block_off = block_vertex_offset[blockIdx.x];

  if (active_edge.x)
    vertices[block_off + vertex_block_id.x - 1] = vertices_local[0];
  if (active_edge.y)
    vertices[block_off + vertex_block_id.y - 1] = vertices_local[1];
  if (active_edge.z)
    vertices[block_off + vertex_block_id.z - 1] = vertices_local[2];

  for (size_t i = 0; i < num_tris; i++) {
    indices[index_offset + block_index_offset[blockIdx.x] + i].x =
        block_off + tris[3 * i] - 1;
    indices[index_offset + block_index_offset[blockIdx.x] + i].y =
        block_off + tris[3 * i + 1] - 1;
    indices[index_offset + block_index_offset[blockIdx.x] + i].z =
        block_off + tris[3 * i + 2] - 1;
  }
}

__global__ void genTriangles::setGlobal() {
  my_block_count3_0 = 0;
  my_block_count3_1 = 0;
}

int2 genTriangles::generateTrisWrapper(hipTextureObject_t tex,
                                       int* activeBlocks, int* numActiveBlocks,
                                       dim3 grid_size3, dim3 block_size3,
                                       dim3 grid_size, int isoVal, uint3 nxyz,
                                       float3** d_vertices_ref,
                                       int3** d_indices_ref) {
  hipMemcpyToSymbol(HIP_SYMBOL(d_isoVal), &isoVal, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(d_neighbourMappingTable), neighbourMappingTable,
                     12 * 4 * sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(d_edgeTable), edgeTable, 256 * sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(d_triTable), triTable, 256 * 16 * sizeof(int));

  // Global offset
  int* d_block_vertex_offset;
  int* d_block_index_offset;

  hipMalloc(&d_block_vertex_offset, (grid_size3.x + 1) * sizeof(int));
  hipMalloc(&d_block_index_offset, (grid_size3.x + 1) * sizeof(int));
  hipMemset(d_block_vertex_offset, 0, (grid_size3.x + 1) * sizeof(int));
  hipMemset(d_block_index_offset, 0, (grid_size3.x + 1) * sizeof(int));

  // store vertices / indices
  float3* d_vertices;
  int3* d_indices;
  hipMalloc(&d_vertices, nxyz.x * nxyz.y * nxyz.z * sizeof(float3));
  hipMalloc(&d_indices, nxyz.x * nxyz.y * nxyz.z * sizeof(int3));
  hipMemset(d_vertices, 0, nxyz.x * nxyz.y * nxyz.z * sizeof(float3));
  hipMemset(d_indices, 0, nxyz.x * nxyz.y * nxyz.z * sizeof(int3));

  setGlobal<<<1, 1>>>();
  hipDeviceSynchronize();

  generateTris<<<grid_size3, block_size3>>>(
      tex, activeBlocks, numActiveBlocks, grid_size, d_block_vertex_offset,
      d_block_index_offset, d_vertices, d_indices);

  int num_vertices = 0, num_indices = 0;
  hipMemcpy(&num_vertices, d_block_vertex_offset + grid_size3.x, sizeof(int),
             hipMemcpyDeviceToHost);
  hipMemcpy(&num_indices, d_block_index_offset + grid_size3.x, sizeof(int),
             hipMemcpyDeviceToHost);

  *d_vertices_ref = d_vertices;
  *d_indices_ref = d_indices;

  hipFree(d_block_vertex_offset);
  hipFree(d_block_index_offset);

  return int2{num_vertices, num_indices};
}
